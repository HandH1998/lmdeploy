#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "src/turbomind/kernels/gemm/gemm.h"
#include "src/turbomind/kernels/gemm/types.h"
#include "src/turbomind/kernels/marlin_qqq_gemm/marlin_qqq_gemm.h"
#include "src/turbomind/models/llama/LlamaLinear.h"
#include "src/turbomind/models/llama/llama_decoder_kernels.h"
#include "src/turbomind/models/llama/llama_params.h"
#include "src/turbomind/models/llama/llama_utils.h"
#include "src/turbomind/utils/allocator.h"
#include <fstream>

namespace turbomind {

template<class T>
struct LlamaLinear<T>::Impl {

    Impl(cublasMMWrapper* cublas_wrapper, hipStream_t stream, IAllocator* allocator):
        cublas_wrapper_(cublas_wrapper), stream_(stream), gemm_s4_s8_(allocator)
    {
        workspace_ = {};

        workspace_.barriers_size = gemm::Gemm::kBarriersSize;
        workspace_.partials_size = gemm::Gemm::kPartialsSize;
        hipMallocAsync(&workspace_.barriers, workspace_.barriers_size, stream_);
        hipMallocAsync(&workspace_.partials, workspace_.partials_size, stream_);
        hipMemsetAsync(workspace_.barriers, 0, workspace_.barriers_size, stream_);
    }

    ~Impl()
    {
        hipFreeAsync(workspace_.barriers, stream_);
        hipFreeAsync(workspace_.partials, stream_);
        workspace_ = {};
    }

    void forward(T*                         output_data,
                 Pitched                    input_data,
                 int8_t*                    quant_input_data,
                 float*                     quant_scale,
                 int                        batch_size,
                 const LlamaDenseWeight<T>& weight,
                 Type                       type      = kGemm,
                 int*                       lora_mask = nullptr)
    {
        if (input_data.pitch == 0) {
            input_data.pitch = weight.input_dims;
        }
        if (lora_mask != nullptr && weight.lora.r > 0) {
            FT_CHECK(type == kGemm);
            // output = lora(x) * scale
            // output = mask(output)
            // output = x*W + output
            cublas_wrapper_->Gemm(HIPBLAS_OP_N,
                                  HIPBLAS_OP_N,
                                  weight.lora.r,                                  // m
                                  batch_size,                                     // n
                                  weight.input_dims,                              // k
                                  (const T*)weight.lora.a,                        // A
                                  weight.lora.r,                                  // lda
                                  input_data.ptr,                                 // B
                                  input_data.pitch,                               // ldb
                                  output_data + batch_size * weight.output_dims,  // C
                                  weight.lora.r);                                 // ldc

            cublas_wrapper_->Gemm(HIPBLAS_OP_N,
                                  HIPBLAS_OP_N,
                                  weight.output_dims,                             // m
                                  batch_size,                                     // n
                                  weight.lora.r,                                  // k
                                  (const T*)weight.lora.b,                        // A
                                  weight.output_dims,                             // lda
                                  output_data + batch_size * weight.output_dims,  // B
                                  weight.lora.r,                                  // ldb
                                  output_data,                                    // C
                                  weight.output_dims,                             // ldc
                                  weight.lora.scale,                              // alpha
                                  0.0f);                                          // beta

            invokeMask(output_data, lora_mask, batch_size, weight.output_dims, stream_);
            sync_check_cuda_error();

            type = kFusedAdd;
        }
        switch (weight.quantization) {
            case QuantMethod::QNone:
                return forwardFp(output_data, input_data, batch_size, weight, type);
            case QuantMethod::AWQ:
            case QuantMethod::GPTQ:
                return forwardInt4(output_data, input_data, batch_size, weight, type);
            case QuantMethod::QQQ:
                return forwardQQQ(output_data, quant_input_data, quant_scale, batch_size, weight, type);
            default:
                FT_CHECK(0);
        }
    }

    void forwardFp(T* output_data, Pitched input_data, int batch_size, const LlamaDenseWeight<T>& weight, Type type)
    {
        cublas_wrapper_->Gemm(HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              weight.output_dims,
                              batch_size,
                              weight.input_dims,
                              (const T*)weight.kernel,
                              weight.output_dims,
                              input_data.ptr,
                              input_data.pitch,
                              output_data,
                              weight.output_dims,
                              1.0f,
                              type == kFusedAdd ? 1.0f : 0.0f);
        sync_check_cuda_error();
    }

    void forwardInt4(T* output_data, Pitched input_data, int batch_size, const LlamaDenseWeight<T>& weight, Type type)
    {
        using namespace gemm;

        const Operation operation{dispatch_policy_,
                                  type == kFusedSiluFfn ? Epilogue::kGatedSilu : Epilogue::kNone,
                                  {QuantType::kNone},
                                  {QuantType::kDefault, weight.group_size},
                                  0,
                                  nullptr};

        const MatrixLayout a_desc{
            get_data_type_v<T>,
            kRowMajor,
            batch_size,
            (int)weight.input_dims,
            input_data.pitch,
        };

        const MatrixLayout c_desc{
            get_data_type_v<T>,
            kRowMajor,
            batch_size,
            (int)weight.output_dims,
            type == kFusedSiluFfn ? (int)weight.output_dims / 2 : (int)weight.output_dims,
        };

        auto ec = gemm_.Run(operation,
                            1.f,
                            input_data.ptr,
                            a_desc,
                            nullptr,
                            {},
                            weight.kernel,
                            weight.k_desc,
                            weight.scales_zeros,
                            weight.q_desc,
                            type == kFusedAdd ? 1.0f : 0.0f,
                            output_data,
                            c_desc,
                            output_data,
                            c_desc,
                            workspace_,
                            stream_);

        if (ec) {
            TM_LOG_ERROR("%s: %d", __PRETTY_FUNCTION__, ec);
            // std::abort();
        }
    }

    // w4a8
    void forwardQQQ(T*                         output_data,
                    const int8_t*              input_data,
                    const float*               act_scale,
                    int                        batch_size,
                    const LlamaDenseWeight<T>& weight,
                    Type                       type)
    {
        // qqq only supports kGemm
        FT_CHECK(type == kGemm);
        if constexpr (std::is_same_v<T, half>) {
            gemm_s4_s8_.Run(output_data,
                            input_data,
                            (const uint*)weight.kernel,
                            act_scale,
                            (const float*)weight.scales_channel,
                            (const half*)weight.scales_zeros,
                            batch_size,
                            weight.output_dims,
                            weight.input_dims,
                            weight.group_size,
                            stream_);
            sync_check_cuda_error();
        }
        else {
            FT_CHECK_WITH_INFO(0, "Not implemented");
        }
    }

    cublasMMWrapper*          cublas_wrapper_;
    gemm::Gemm                gemm_;
    gemm::DispatchPolicy      dispatch_policy_{gemm::DispatchPolicy::kDefault};
    marlin_qqq::MarlinQQQGemm gemm_s4_s8_;
    hipStream_t              stream_{};

    gemm::Workspace workspace_;
};

template<class T>
LlamaLinear<T>::LlamaLinear(cublasMMWrapper* cublas_wrapper, hipStream_t stream, IAllocator* allocator):
    impl_{std::make_shared<Impl>(cublas_wrapper, stream, allocator)}
{
}

template<class T>
void LlamaLinear<T>::forward(T*                         output_data,
                             Pitched                    input_data,
                             int8_t*                    quant_input_data,
                             float*                     quant_scale,
                             int                        batch_size,
                             const LlamaDenseWeight<T>& weight,
                             Type                       type,
                             int*                       lora_mask)
{
    impl_->forward(output_data, input_data, quant_input_data, quant_scale, batch_size, weight, type, lora_mask);
}

template<class T>
void LlamaLinear<T>::set_measure(bool measure)
{
    impl_->dispatch_policy_ = measure ? gemm::DispatchPolicy::kMeasure : gemm::DispatchPolicy::kReuse;
}

template<class T>
int LlamaLinear<T>::Export(std::ostream& os)
{
    if (os) {
        return impl_->gemm_.Export(os);
    }
    return 0;
}

template<class T>
int LlamaLinear<T>::Import(std::istream& is)
{
    auto n_records = 0;
    if (is) {
        n_records = impl_->gemm_.Import(is);
    }
    if (n_records) {
        impl_->dispatch_policy_ = gemm::DispatchPolicy::kReuse;
    };
    return n_records;
}

template<class T>
std::pair<int*, int*> LlamaLinear<T>::getQQQBuffer()
{
    return impl_->gemm_s4_s8_.getBuffer();
}

template<class T>
void LlamaLinear<T>::setQQQBuffer(int* reduce_buf, int* workspace_buf)
{
    impl_->gemm_s4_s8_.setBuffer(reduce_buf, workspace_buf);
}

template<class T>
std::vector<int> LlamaLinear<T>::GetTuningSeq() const
{
    return impl_->gemm_.GetTuningSeq();
}

#ifdef ENABLE_FP32
template class LlamaLinear<float>;
#endif
template class LlamaLinear<half>;
#ifdef ENABLE_BF16
template class LlamaLinear<__hip_bfloat16>;
#endif

}  // namespace turbomind
